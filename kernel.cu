#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdio>

#define CUDA_CHECK(expr)                                                \
  do {                                                                  \
    hipError_t err = (expr);                                           \
    if (err != hipSuccess) {                                           \
      fprintf(stderr, "CUDA Error Code : %d\n     Error String: %s\n", \
              err, hipGetErrorString(err));                            \
      exit(err);                                                        \
    }                                                                   \
  } while (0)

__global__ void kernel() {
  printf("cuda kernel called!\n");
}

void launch() {
  int driver_version;
  CUDA_CHECK(hipDriverGetVersion(&driver_version));
  printf("CUDA driver version: %d\n", driver_version);
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}
